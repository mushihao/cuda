#include "hip/hip_runtime.h"
/****************************************************************************************
An example of multiple producer multiple consumer test
The while loop performs two tasks:
1. Producing: each thread reads the source array, find elements greater than the threshold 
              value, then put the array index into the destination array
2. Consuming: each thread reads the array index from the destination array and then update
              the threashold value
Notes: A simplification here is the termination condition, because the number of elements
      greater than the threshold value is known. 
*****************************************************************************************/
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <vector>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

constexpr int BLOCK_SIZE = 32;
constexpr int NUM_BLOCK = 7;
void swap(int* a, int* b) {
    int temp = *a;
    *a = *b;
    *b = temp;
    return;
}
__global__ void mpmc(volatile int* src, volatile int* dst, int* dst_read_counter, int* dst_write_counter, int array_size, int threshold) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    bool src_process_done = false;
    int offset = 0;
    while (true) {
        if (!src_process_done) {
            int pos = 0;
            int id = tid + offset;
            if (id >= array_size) {
                src_process_done = true;
            }
            else {
                int32_t element = src[id];
                if (element > threshold) {
                    pos = atomicAdd(dst_write_counter, 1);
                    dst[pos] = id;
                    __threadfence();
                }
                offset += BLOCK_SIZE * NUM_BLOCK;
                continue;
            }
        }
        
        int old_read_val = atomicAdd(dst_read_counter, 1);
        int old_write_val = atomicAdd(dst_write_counter, 0);
        if (old_read_val >= old_write_val) {
            atomicAdd(dst_read_counter, -1);
        }
        else {
            int idx = dst[old_read_val];
            if (src[idx] > threshold) {
                src[idx] = 0;
            }
            else {
                src[idx] = 3;
            }
            dst[old_read_val] = array_size;
        }
        if (src_process_done && (old_read_val == old_write_val)) {
            break;
        }
        
    }
}


int main() {
    const int length = 1024 * 1024;
    int threshold_value = length - 108;
    int* src = new int[length];
    int* dst = new int[length];
    int valid_value = 0;
    for (int i = 0; i < length; i++) {
        src[i] = i;
        dst[i] = 0;
    }
    for (int i = length - 1; i > 0; --i) {
        int j = std::rand() % (i + 1);
        swap(&src[i], &src[j]);
    }
    std::cout << " Done Initilization\n";
    int init_value = 0;
    int result;
    int* device_src;
    int* device_dst;
    int* dst_read_counter;
    int* dst_write_counter;
    hipMalloc(&device_src, length * sizeof(int));
    hipMalloc(&device_dst, length * sizeof(int));
    hipMalloc(&dst_read_counter, sizeof(int));
    hipMalloc(&dst_write_counter, sizeof(int));
    hipMemcpy(device_src, src, length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_dst, dst, length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dst_read_counter, &init_value, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dst_write_counter, &init_value, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    mpmc << <NUM_BLOCK, BLOCK_SIZE >> > (device_src, device_dst, dst_read_counter, dst_write_counter, length, threshold_value);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(dst, device_src, length * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&result, dst_read_counter, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Run time is " << time << " ms." << std::endl;
    valid_value = length - threshold_value - 1;
    bool error = false;
    if (result != valid_value) {
        std::cout << " read result is wrong, expected: " << valid_value << " , result: " << result << std::endl;
        error = true;
    }
    hipMemcpy(&result, dst_write_counter, sizeof(int), hipMemcpyDeviceToHost);
    if (result != valid_value) {
        std::cout << " write result is wrong, expected: " << valid_value << " , result: " << result << std::endl;
        error = true;
    }
    
    for (int i = 0; i < length; i++) {
        if (dst[i] > threshold_value) { //% 3 != 0 ) {
            std::cout << " origin value is wrong: i " << i << " value: " << dst[i] << std::endl;
            error = true;
        }
    }
    hipMemcpy(dst, device_dst, length * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < result; i++) {
        if (dst[i]!= length) {
            std::cout << " dst is wrong: i " << i << " value: " << dst[i] << std::endl;
            error = true;
        }
    }

    hipFree(device_src);
    hipFree(device_dst);
    hipFree(dst_read_counter);
    hipFree(dst_write_counter);
    delete[] src;
    delete[] dst;
    if (error) {
        return 1;
    }
    else {
        return 0;
    }
} 
