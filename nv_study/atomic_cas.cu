#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <vector>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

constexpr int BLOCK_SIZE = 32;
constexpr int NUM_BLOCK = 2;
template<typename T>
__global__ void test_kernel(T* src, volatile int* dst) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    atomicCAS(src + tid, 1, 1);
}


int main() {
    const int length = 16 * 1024;
    const int loop_cnt = 1;
    int* src = new int[length];
    int* dst = new int[length];
    int valid_value = 0;
    for (int i = 0; i < length; i++) {
        src[i] = i;
        dst[i] = 0;
    }

    std::cout << " Done Initilization\n";
    int* device_src;
    int* device_dst;
    hipMalloc(&device_src, length * sizeof(int));
    hipMalloc(&device_dst, length * sizeof(int));
    hipMemcpy(device_src, src, length * sizeof(int), hipMemcpyHostToDevice);
    //  hipMemcpy(device_dst, dst, length * sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    test_kernel<int> << <NUM_BLOCK, BLOCK_SIZE >> > (reinterpret_cast<int*>(device_src), device_dst);
    test_kernel<unsigned long long int> << <NUM_BLOCK, BLOCK_SIZE >> > (reinterpret_cast<unsigned long long int*>(device_src), device_dst);
    test_kernel<unsigned short> << <NUM_BLOCK, BLOCK_SIZE >> > (reinterpret_cast<unsigned short*>(device_src), device_dst);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(dst, device_dst, length * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Run time is " << time / loop_cnt << " ms." << std::endl;
    hipFree(device_src);
    hipFree(device_dst);
    delete[] src;
    delete[] dst;
    return 0;
}
